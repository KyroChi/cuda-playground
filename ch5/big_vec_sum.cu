#include "hip/hip_runtime.h"
/**
 * Do GPU compute on more blocks / threads than are available.
 */

#include "../common/book.h"

#define N ( 512 * 1024 )

void
add ( int* a, int* b, int* c )
{
	int tid = 0;
	while ( tid < N ) {
		c[tid] = a[tid] + b[tid];
		tid += 1;
	}
}

__global__ void
add_cuda ( int* a, int* b, int* c )
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while ( tid < N ) {
		c[tid] = a[tid] + b[tid];
		tid += blockDim.x * gridDim.x;
	}
}

int
main ( void )
{
	struct timespec start, finish;
	double elapsed;
	
	int a[N], b[N], c[N];

	for ( int ii = 0; ii < N; ii++ ) {
		a[ii] = -ii;
		b[ii] = ii * ii;
	}

	clock_gettime(CLOCK_MONOTONIC, &start);
	add(a, b, c);
	clock_gettime(CLOCK_MONOTONIC, &finish);

	elapsed = (finish.tv_sec - start.tv_sec);
	elapsed += (finish.tv_nsec - start.tv_nsec) / 1000000000.0;

	printf("Host took \t%.8f seconds\n", elapsed);

	int *dev_a, *dev_b, *dev_c;

	clock_gettime(CLOCK_MONOTONIC, &start);
	
	HANDLE_ERROR( hipMalloc( (void**)&dev_a,
				  N * sizeof(int) ) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_b,
				  N * sizeof(int) ) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_c,
				  N * sizeof(int) ) );

	// Copy a and b into device memory
	HANDLE_ERROR( hipMemcpy( dev_a, a, N * sizeof(int),
				  hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy( dev_b, b, N * sizeof(int),
				  hipMemcpyHostToDevice ) );

	add_cuda<<<(N+127)/128, 256>>>( dev_a, dev_b, dev_c );

	HANDLE_ERROR( hipMemcpy( c, dev_c, N * sizeof(int),
				  hipMemcpyDeviceToHost ) );

	clock_gettime(CLOCK_MONOTONIC, &finish);

	elapsed = (finish.tv_sec - start.tv_sec);
	elapsed += (finish.tv_nsec - start.tv_nsec) / 1000000000.0;

	printf("Device took \t%.8f seconds\n", elapsed);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	
	return 0;
}