#include "hip/hip_runtime.h"
#include "../common/book.h"

#define N 10

__global__ void
add ( int* a, int* b, int* c )
{
	int tid = threadIdx.x;
	if ( tid < N ) {
		c[tid] = a[tid] + b[tid];
	}
}

int
main ( void )
{
	struct timespec start, finish;
	double elapsed;
	
	int a[N], b[N], c[N];

	for ( int ii = 0; ii < N; ii++ ) {
		a[ii] = -ii;
		b[ii] = ii * ii;
	}

	int *dev_a, *dev_b, *dev_c;

	clock_gettime(CLOCK_MONOTONIC, &start);
	
	HANDLE_ERROR( hipMalloc( (void**)&dev_a,
				  N * sizeof(int) ) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_b,
				  N * sizeof(int) ) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_c,
				  N * sizeof(int) ) );

	// Copy a and b into device memory
	HANDLE_ERROR( hipMemcpy( dev_a, a, N * sizeof(int),
				  hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy( dev_b, b, N * sizeof(int),
				  hipMemcpyHostToDevice ) );

	add<<<(N+127)/128, 128>>>( dev_a, dev_b, dev_c );

	HANDLE_ERROR( hipMemcpy( c, dev_c, N * sizeof(int),
				  hipMemcpyDeviceToHost ) );

	clock_gettime(CLOCK_MONOTONIC, &finish);

	elapsed = (finish.tv_sec - start.tv_sec);
	elapsed += (finish.tv_nsec - start.tv_nsec) / 1000000000.0;

	printf("Device took \t%.8f seconds\n", elapsed);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	
	return 0;
}